#include "hip/hip_runtime.h"

#include "kernel.cuh"


__device__ int mandelbrot(TypePrecision x, TypePrecision y, TypePrecision pr, TypePrecision pi, int max_iter, TypePrecision cutoff) {
    TypePrecision zr = pr;
    TypePrecision zi = pi;

    TypePrecision cr = x;
    TypePrecision ci = y;
    int iter = 0;
    while (zr * zr + zi * zi <= cutoff && iter < max_iter) {
        TypePrecision temp = zr * zr - zi * zi + cr;
        zi = 2.0 * zr * zi + ci;
        zr = temp;
        iter++;
    }
    return iter;
}


__device__ int julia(TypePrecision x, TypePrecision y, TypePrecision pr, TypePrecision pi, int max_iter, TypePrecision cutoff) {

    TypePrecision zr = x;
    TypePrecision zi = y;

    TypePrecision cr = pr;
    TypePrecision ci = pi;
    int iter = 0;
    while (zr * zr + zi * zi <= cutoff && iter < max_iter) {
        TypePrecision temp = zr * zr - zi * zi + cr;
        zi = 2.0 * zr * zi + ci;
        zr = temp;
        iter++;
    }
    return iter;
}


__global__ void mandelbrot_kernel(Color* image,
    TypePrecision xMin, TypePrecision xMax, TypePrecision yMin, 
    TypePrecision yMax, int max_iter, TypePrecision cutoff, FRACTAL_TYPES frac_type, 
    TypePrecision par_r, TypePrecision par_i) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= CANVAS_SIZE_X || j >= CANVAS_SIZE_Y) return;

    TypePrecision x0 = xMin + (xMax - xMin) * i / CANVAS_SIZE_X;
    TypePrecision y0 = yMin + (yMax - yMin) * j / CANVAS_SIZE_Y;

    int iter; 
    if (frac_type == MANDELBROT) {
        iter = mandelbrot(x0, y0, par_r, par_i, max_iter, cutoff);
    }
    else //frac_type == JULIA
    {
        iter = julia(x0, y0, par_r, par_i, max_iter, cutoff);
    }
    
    int color = (iter == max_iter) ? 0 : (255 * iter / max_iter);

    Color pixelColor{};
    pixelColor.red = color;
    pixelColor.green = color;
    pixelColor.blue = color;

    int idx = j * CANVAS_SIZE_X + i;
    image[idx] = pixelColor;
}


void calcurate_fractal_cuda(Color* frame) {
    Color* device_frame;

    SettingsSingleton settings = SettingsSingleton::getInstance();
    int max_iter = settings.number_of_iterations;
    Bounds bounds = settings.getBounds();
    size_t img_size = CANVAS_SIZE_X * CANVAS_SIZE_Y * sizeof(Color);


    hipMalloc(&device_frame, img_size);

    dim3 blockSize(16, 16);
    dim3 gridSize((CANVAS_SIZE_X + blockSize.x - 1) / blockSize.x, (CANVAS_SIZE_Y + blockSize.y - 1) / blockSize.y);


    
#pragma warning( push )
#pragma warning( disable : E0029)
    // Linter error 
    mandelbrot_kernel <<<gridSize, blockSize >>> (device_frame, bounds.left_x,
        bounds.right_x, bounds.bottom_y, bounds.top_y,
        max_iter, settings.cut_off_value, settings.fractal_type, settings.real_part_parameter, settings.im_part_parameter);
#pragma warning( pop )

    hipMemcpy(frame, device_frame, img_size, hipMemcpyDeviceToHost);

    hipFree(device_frame);

}




